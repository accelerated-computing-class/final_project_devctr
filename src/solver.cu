#include "hip/hip_runtime.h"
#include <algorithm>
#include <sstream>
#include <type_traits>
#ifdef CLANG
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_runtime_wrapper.h>

#endif

#include "fmt/format.h"
#include <cstddef>
#include <cstdint>
#include <cub/block/block_reduce.cuh>
#include <hipcub/hipcub.hpp>
#include <cuda/std/array>
#include <cuda/std/tuple>
#include <fmt/base.h>
#include <fmt/core.h>
#include <fmt/ranges.h>
#include <limits>
#include <tuple>
#include <vector>

#define STR_DETAIL(x) #x
#define STR(x) STR_DETAIL(x)

#define cuda_error(code)                                                       \
  {                                                                            \
    if ((code) != hipSuccess) [[unlikely]] {                                  \
      fprintf(stderr,                                                          \
              "cuda error %s in file " __FILE__ ":" STR(__LINE__) "\n",        \
              hipGetErrorString(code));                                       \
      if (abort)                                                               \
        throw std::runtime_error("assertion failed");                          \
      ;                                                                        \
    }                                                                          \
  }

template <typename T> using pair = cuda::std::array<T, 2>;

template <typename T> T constexpr cdiv(T a, T b) { return (a + b - 1) / b; }

template <unsigned int n> struct bitset {
  uint32_t data[cdiv(n, 32u)];

  template <typename T> bool constexpr get(T i) const {
    return (data[i / 32u] >> (i % 32)) & 1;
  }
  template <typename T> void constexpr set(T i, bool b) {
    if (b)
      data[i / 32u] |= 1u << (i % 32u);
    else
      data[i / 32u] &= ~(1u << (i % 32u));
  }

  static bitset<n> __host__ full(bool b) {
    bitset<n> res;
    for (int i = 0; i < cdiv(n, 32u); i++) {
      res.data[i] = b ? ~0u : 0u;
    }
    return res;
  }
  static bitset<n> constexpr from(cuda::std::array<bool, n> const &arr) { 
    bitset<n> res;
    for (int i = 0; i < cdiv(n, 32u); i++) {
      res.data[i] = 0;
    }
    for (int i = 0; i < n; i++) {
      res.set(i, arr[i]);
    }
    return res;
  }
};
template <typename T, int x, int y> struct range_array {
  cuda::std::array<int, x + 1> idx;
  cuda::std::array<T, y> val;

  auto static from_tiplet(
      cuda::std::array<cuda::std::tuple<int, T>, y> const &triplet) {
    cuda::std::array<int, x + 1> delta;
    range_array res;

    for (auto &i : delta)
      i = 0;

    for (auto &i : res.idx)
      i = 0;

    for (auto &[i, _] : triplet)
      ++res.idx[i + 1];

    for (int i = 0; i < x; i++)
      res.idx[i + 1] += res.idx[i];

    for (auto &[i, j] : triplet) {
      res.val[res.idx[i] + delta[i]] = j;
      ++delta[i];
    }
    for (int i = 0; i < x; ++i)
      if (res.idx[i] + delta[i] != res.idx[i + 1]) {
        fmt::println("delta: {}", fmt::join(delta, ", "));
        fmt::println("idx: {}", fmt::join(res.idx, ", "));
        fmt::println("triplet: {}", fmt::join(triplet, ", "));

        throw std::runtime_error("range array construction failed");
      }

    for (int i = 0; i < x; ++i) {
      std::stable_sort(res.val.begin() + res.idx[i],
                       res.val.begin() + res.idx[i + 1]);
    }
    return res;
  }
};

template <int n, int m, int n_terms> struct problem_t {
  auto static constexpr n_var = n;
  auto static constexpr n_constr = m;
  cuda::std::array<int, n> obj;
  cuda::std::array<int, m> rhs;
  cuda::std::array<int, m> rhs_n;
  bitset<m> is_eq;
  // first item is constraint index, second item is coefficient
  range_array<pair<int>, n, n_terms> var_2_constr;
  // first item is variable index, second item is coefficient
  range_array<pair<int>, m, n_terms> constr_2_var;
};

template <int n, int m, int n_terms>
struct fmt::formatter<problem_t<n, m, n_terms>> {
  template <typename ParseContext> constexpr auto parse(ParseContext &ctx) {
    return ctx.begin();
  }

  auto format(problem_t<n, m, n_terms> const &t, format_context &ctx) const
      -> format_context::iterator {
    cuda::std::array<cuda::std::array<int, n>, m> A;
    for (auto &c : A) {
      for (auto &x : c) {
        x = 0;
      }
    }
    for (int var_idx = 0; var_idx < n; var_idx++) {
      for (int j = t.var_2_constr.idx[var_idx];
           j < t.var_2_constr.idx[var_idx + 1]; j++) {
        auto const [constr_idx, coeff] = t.var_2_constr.val[j];
        if (A[constr_idx][var_idx] != 0)
          throw std::runtime_error("duplicate variable");
        A[constr_idx][var_idx] = coeff;
      }
    }

    std::stringstream ss;
    ss << fmt::format("problem_t<{}, {}, {}>\n", n, m, n_terms);
    ss << "min ";
    for (int i = 0; i < n; i++) {
      ss << t.obj[i] << "x_" << i;
      if (i != n - 1)
        ss << " + ";
    }

    ss << "\n";

    for (int constr_idx = 0; constr_idx < m; constr_idx++) {
      bool not_first = false;
      for (int var_idx = 0; var_idx < n; var_idx++) {
        if (A[constr_idx][var_idx] == 0)
          continue;
        if (not_first)
          ss << " + ";
        if (A[constr_idx][var_idx] != 1)
          ss << A[constr_idx][var_idx];
        ss << "x_" << var_idx;
        not_first = true;
      }
      if (t.is_eq.get(constr_idx))
        ss << " = ";
      else
        ss << " <= ";
      ss << t.rhs[constr_idx] << "\n";
    }

    return fmt::format_to(ctx.out(), "{}", ss.str());
  }
};

template <int n_var, int n_constr> struct alignas(8) solution_t {
  int index;
  int upper_bound;
  int obj;
  bitset<n_var * 2> var;

  cuda::std::array<int, n_constr> rhs;
  cuda::std::array<int, n_constr> rhs_n;
};
// template <typename T> union data_u {
//   static_assert(sizeof(T) % 4 == 0);
//   T x;
//   uint32_t data[sizeof(T) / 32];
// };

template <int n, int m> struct fmt::formatter<solution_t<n, m>> {
  template <typename ParseContext> constexpr auto parse(ParseContext &ctx) {
    return ctx.begin();
  }

  auto format(solution_t<n, m> const &t, format_context &ctx) const
      -> format_context::iterator {
    std::stringstream ss;
    ss << fmt::format("solution_t<{}, {}>\n", n, m);
    ss << "index: " << t.index << "\n";
    ss << "upper_bound: " << t.upper_bound << "\n";
    ss << "obj: " << t.obj << "\n";
    ss << fmt::format("rhs: {}\n", t.rhs);
    ss << fmt::format("rhs_n: {}\n", t.rhs_n);

    ss << "var: \n";
    for (int i = 0; i < n; i++) {
      bool can_be_zero = t.var.get(i * 2);
      bool can_be_one = t.var.get(i * 2 + 1);
      if (can_be_zero && can_be_one)
        ss << "x_" << i << " = 0, 1";
      else if (can_be_zero)
        ss << "x_" << i << " = 0";
      else if (can_be_one)
        ss << "x_" << i << " = 1";
      else
        ss << "x_" << i << " = inf";
      ss << "\n";
    }
    return fmt::format_to(ctx.out(), "{}", ss.str());
  }
};

template <typename T> __device__ void mycpy(T const *src, T *dst) {

  using I = uint32_t;
  static_assert(sizeof(T) % sizeof(I) == 0);
  I *Isrc = reinterpret_cast<I *>((void *)src);
  I *Idst = reinterpret_cast<I *>((void *)dst);
  for (int i = threadIdx.x; i < sizeof(T) / sizeof(I); i += blockDim.x) {
    Idst[i] = Isrc[i];
  }
}

template <int n_threads, int n_var, int n_constr, int n_terms>
__global__ void
traverse(problem_t<n_var, n_constr, n_terms> const *const problem,
         solution_t<n_var, n_constr> const *const queue, // SDF
         uint32_t *delta_mask,                           // SDF
         solution_t<n_var, n_constr> *delta_queue) {
  using sol_t = solution_t<n_var, n_constr>;
  sol_t const *__restrict__ const qel = queue + blockIdx.x;
  __shared__ bool kill_switch;
  __shared__ sol_t cur;
  __shared__ sol_t next;

  mycpy(qel, &cur);
  __syncthreads();
  if (cur.index >= problem->n_var) {
    if (threadIdx.x <= 1)
      delta_mask[2 * blockIdx.x + threadIdx.x] = 0;
    return;
  }

  for (auto val : {false, true}) {
    __syncthreads();
    auto const dqidx = 2 * blockIdx.x + val;
    kill_switch = false;
    if (!cur.var.get(cur.index * 2 + val)) {
      if (threadIdx.x == 0)
        delta_mask[dqidx] = 0;

      continue;
    }
    __syncthreads();
    mycpy(&cur, &next);
    __syncthreads();
    if (threadIdx.x == 0)
      next.var.set(cur.index * 2, val == false);
    if (threadIdx.x == 1)
      next.var.set(cur.index * 2 + 1, val == true);
    if (threadIdx.x == 2)
      next.index++;
    __syncthreads();
    auto const var_idx = cur.index;
    auto const var_begin = problem->var_2_constr.idx[var_idx];
    auto const var_end = problem->var_2_constr.idx[var_idx + 1];
    for (uint32_t i = threadIdx.x + var_begin; i < var_end && !kill_switch;
         i += blockDim.x) {
      auto const [constr_idx, coeff] = problem->var_2_constr.val[i];
      next.rhs[constr_idx] -= coeff * val;
      ++next.rhs_n[constr_idx];
      auto const rhs_n = next.rhs_n[constr_idx];
      auto const rhs_exp = problem->rhs_n[constr_idx];
      auto const is_eq = problem->is_eq.get(constr_idx);
      if (rhs_n == rhs_exp) {
        if ((is_eq && next.rhs[constr_idx] != 0) ||
            (!is_eq && next.rhs[constr_idx] > 0)) {
          kill_switch = true;
          break;
        }
      }
    }

    __syncthreads();
    if (threadIdx.x == 0) {
      if (kill_switch)
        next.obj = std::numeric_limits<int>::max();
      else
        next.obj += problem->obj[cur.index] * val;
    }
    if (threadIdx.x == 0) {
      printf("blockidx %d, val %d, kill_switch %d dqidx %d\n", blockIdx.x, val,
             kill_switch, dqidx);
    }

    if (threadIdx.x == 0) {
      delta_mask[dqidx] = !kill_switch && (next.index < n_var);
    }
    mycpy(&next, delta_queue + dqidx);

    __syncthreads();
  }
}

template <uint32_t n_threads, int n_var, int n_constr>
__global__ void push_back(uint32_t *delta_cumsum,
                          solution_t<n_var, n_constr> const *const delta_queue,
                          solution_t<n_var, n_constr> *queue) {
  using sol_t = solution_t<n_var, n_constr>;
  auto const cur_idx = delta_cumsum[blockIdx.x];
  auto const prev_idx = blockIdx.x == 0 ? 0 : delta_cumsum[blockIdx.x - 1];

  if (cur_idx == prev_idx) {
    return;
  }
  mycpy(&delta_queue[blockIdx.x], &queue[cur_idx - 1]);
}

template <typename T> __device__ T broadcast(T const &x) {
  __shared__ T s;
  if (threadIdx.x == 0)
    s = x;
  __syncthreads();
  return s;
}
template <uint32_t n_threads, int n_var, int n_constr>
__global__ void
update_bounds(solution_t<n_var, n_constr> const *const delta_queue,
              uint32_t *delta_mask, uint32_t delta_q_size,
              solution_t<n_var, n_constr> *best) {
  uint32_t best_idx = 0;
  int best_val = std::numeric_limits<int>::max();
  for (int i = threadIdx.x; i < delta_q_size; i += blockDim.x) {
    printf("delta_queue[%d].index %d obj %d\n", i, delta_queue[i].index,
           delta_queue[i].obj);
    if (delta_queue[i].index == n_var && delta_queue[i].obj < best_val) {
      best_idx = i;
      best_val = delta_queue[i].obj;
    }
  }
  if (best_idx != 0 || best_val != std::numeric_limits<int>::max())
    printf("threadIdx.x %d, best_idx %d best_val %d\n", threadIdx.x, best_idx,
           best_val);
  cuda::std::pair<int, uint32_t> candidate {best_val, best_idx};
  using BlockReduce =
      hipcub::BlockReduce<cuda::std::pair<int, uint32_t> , n_threads>;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  auto [r_best_val, r_best_idx] =
      broadcast(BlockReduce(temp_storage)
                    .Reduce(candidate,
                            [](auto const &a, auto const &b) {
                              return a.first < b.first ? a : b;
                            }));
  if (threadIdx.x == 0)
    printf("r_best_val %d, r_best_idx %d\n", r_best_val, r_best_idx);
  if (r_best_val < best->obj) {
    mycpy(&delta_queue[r_best_idx], best);
  }
}
int main() {
  /*
max x + 2y + 3z + 4w
s.t.
x + y = 1
z + w = 1
*/
  cuda::std::array<cuda::std::tuple<int, pair<int>>, 4> const var_2_constr{
      cuda::std::make_tuple(0, pair{0, 1}),
      cuda::std::make_tuple(1, pair{0, 1}),
      cuda::std::make_tuple(2, pair{1, 1}),
      cuda::std::make_tuple(3, pair{1, 1})};

  cuda::std::array<cuda::std::tuple<int, pair<int>>, 4> const constr_2_var{
      cuda::std::make_tuple(0, pair{0, 1}),
      cuda::std::make_tuple(0, pair{1, 1}),
      cuda::std::make_tuple(1, pair{2, 1}),
      cuda::std::make_tuple(1, pair{3, 1})};
  auto const problem = problem_t<4, 2, 4>{
      .obj = {1, 2, 3, 4},
      .rhs = {1, 1},
      .rhs_n = {2, 2},
      .is_eq = bitset<2>::full(true),
      .var_2_constr = range_array<pair<int>, 4, 4>::from_tiplet(var_2_constr),
      .constr_2_var = range_array<pair<int>, 2, 4>::from_tiplet(constr_2_var),
  };

  decltype(problem) *cuda_prob = nullptr;
  cuda_error(hipMalloc((void **)&cuda_prob, sizeof(problem)));
  cuda_error(hipMemcpy((void *)cuda_prob, &problem, sizeof(problem),
                        hipMemcpyHostToDevice));
  solution_t<4, 2> init_sol{.index = 0,
                            .upper_bound = std::numeric_limits<int32_t>::max(),
                            .obj = 0,
                            .var = bitset<8>::full(true),
                            .rhs = problem.rhs,
                            .rhs_n = {0, 0}};
  constexpr auto n_blocks = 1024;
  constexpr auto n_threads = 64;
  constexpr auto n_outcomes = 2;
  decltype(init_sol) *queue = nullptr;
  auto const q_max_size = n_blocks * decltype(problem)::n_var;
  std::vector<decltype(init_sol)> cpu_queue(q_max_size);
  cuda_error(hipMalloc((void **)&queue, sizeof(init_sol) * q_max_size));
  cuda_error(hipMemcpy((void *)queue, &init_sol, sizeof(init_sol),
                        hipMemcpyHostToDevice));

  decltype(init_sol) *best_solution = nullptr;
  {
    cuda_error(hipMalloc((void **)&best_solution, sizeof(init_sol)));
    auto cpu_best_sol = init_sol;
    cpu_best_sol.obj = std::numeric_limits<int>::max();
    cuda_error(hipMemcpy(best_solution, &cpu_best_sol, sizeof(init_sol),
                          hipMemcpyHostToDevice));
  }

  decltype(init_sol) *delta_queue = nullptr;
  cuda_error(hipMalloc((void **)&delta_queue,
                        sizeof(init_sol) * n_blocks * n_outcomes));

  uint32_t *delta_mask = nullptr;
  uint32_t *delta_cumsum = nullptr;
  cuda_error(hipMalloc((void **)&delta_mask,
                        sizeof(uint32_t) * n_blocks * n_outcomes));

  cuda_error(hipMalloc((void **)&delta_cumsum,
                        sizeof(uint32_t) * n_blocks * n_outcomes));
  cuda_error(
      hipMemset(delta_mask, 0, sizeof(uint32_t) * n_blocks * n_outcomes));
  cuda_error(
      hipMemset(delta_cumsum, 0, sizeof(uint32_t) * n_blocks * n_outcomes));
  auto q_size = 1;

  void *d_temp_storage = nullptr;
  size_t temp_storage_bytes = 0;
  hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, delta_mask,
                                delta_cumsum, n_blocks * n_outcomes);
  cuda_error(hipMalloc(&d_temp_storage, temp_storage_bytes));

  fmt::println("launching kernel for problem \n{}", problem);
  fmt::println("initial solution \n{}", init_sol);
  int itermax = 20;
  std::vector<uint32_t> cpu_delta_mask(n_blocks * n_outcomes);
  while (q_size > 0) {
    auto const n_blocks_l = std::min(q_size, n_blocks);
    fmt::println("-----------------------------\nthere {} jobs in the queue, "
                 "launching {}",
                 q_size, n_blocks_l);
    traverse<n_threads><<<n_blocks_l, n_threads>>>(
        cuda_prob, queue + q_size - n_blocks_l, delta_mask, delta_queue);
    q_size -= n_blocks_l;

    update_bounds<1024><<<1, 1024>>>(delta_queue, delta_mask,
                                     n_blocks_l * n_outcomes, best_solution);

    cpu_delta_mask.resize(n_blocks_l * n_outcomes);
    cuda_error(hipMemcpy(cpu_delta_mask.data(), delta_mask,
                          sizeof(uint32_t) * n_blocks_l * n_outcomes,
                          hipMemcpyDeviceToHost));
    fmt::println("delta_mask: {}", fmt::join(cpu_delta_mask, ", "));
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes,
                                  delta_mask, delta_cumsum,
                                  n_blocks_l * n_outcomes);
    push_back<n_threads><<<n_blocks_l * n_outcomes, n_threads>>>(
        delta_cumsum, delta_queue, queue);
    uint32_t q_detla = 0;
    cuda_error(hipMemcpy(&q_detla, delta_cumsum + n_blocks_l * n_outcomes - 1,
                          sizeof(uint32_t), hipMemcpyDeviceToHost));
    fmt::println("q_delta: {}", q_detla);
    q_size += q_detla;
    cuda_error(hipMemcpy(cpu_queue.data(), queue, sizeof(init_sol) * q_size,
                          hipMemcpyDeviceToHost));
    for (int i = 0; i < q_size; i++) {
      fmt::println("queue[{}]: \n{}", i, cpu_queue[i]);
    }
    {
      decltype(init_sol) cpu_best_sol;
      cuda_error(hipMemcpy(&cpu_best_sol, best_solution, sizeof(init_sol),
                            hipMemcpyDeviceToHost));
      fmt::println("best solution: \n{}", cpu_best_sol);
    }
    if (--itermax == 0)
      break;
  }
};
